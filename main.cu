#include "hip/hip_runtime.h"
#include <sstream>
#include <fstream>
#include <iostream>
#include <time.h>
#include <string>
#include "RayTracer.h"
#define X_BLOCK_SIZE 8
#define Y_BLOCK_SIZE 8

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line);
__global__ void render(vec3* frame_buffer, int max_x, int max_y);

int main(){
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    int driver_version = 0, runtime_version = 0;
    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
				prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
        	    prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
				2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Driver Version: %d\n" 
	       "  Runtime Version: %d\n",
	       driver_version, runtime_version);
    }
	float s = 0.58f;
	for(int i = 0; i < 20; i++){
		s = s + .021;
		char buffer [50];
		sprintf (buffer, "image%u.ppm", i);
		std::string fp(buffer);
		std::cout << "  Filepath: " << fp << "\n";
		std::cout << "  squirc: " << s << "\n";
		RayTracer RenderedImage(640,360,100,X_BLOCK_SIZE,Y_BLOCK_SIZE,s);
		
		RenderedImage.render_image();
		RenderedImage.write_image(fp);
		
		std::cout << "  image: " << i << " complete." << "\n";
	}
    return 0;
}
